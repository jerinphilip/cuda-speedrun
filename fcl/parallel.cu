#include <hip/hip_runtime.h>

#include <cstdio>
#define N 100

__global__ void vsqr(int *A) {
  // Kernel computes vsqr for one  data-item.
  A[threadIdx.x] = threadIdx.x * threadIdx.x;
}

int main() {
  int host_buffer[N];

  int *device_buffer;
  hipMalloc(&device_buffer, N * sizeof(int));

  vsqr<<<1, N>>>(device_buffer);
  hipMemcpy(host_buffer, device_buffer, N * sizeof(int),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  for (int i = 0; i < N; i++) {  // NOLINT
    printf("%d\n", host_buffer[i]);
  }
  return 0;
}
