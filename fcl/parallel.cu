#include <hip/hip_runtime.h>

#include <cstdio>

#include "kernels.h"

#define N 100

int main() {
  int host_buffer[N];

  int *device_buffer;
  hipMalloc(&device_buffer, N * sizeof(int));

  vsqr<<<1, N>>>(device_buffer);
  hipMemcpy(host_buffer, device_buffer, N * sizeof(int),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  for (int i = 0; i < N; i++) {  // NOLINT
    printf("%d\n", host_buffer[i]);
  }
  return 0;
}
