#include <hip/hip_runtime.h>

#include <cstdio>

using dim_t = int32_t;

__global__ void vsqr(int *A) {
  // Kernel computes vsqr for one  data-item.
  A[threadIdx.x] = threadIdx.x * threadIdx.x;
}

__global__ void vsqr_(int *A) {  // NOLINT
  // Kernel computes vsqr for one  data-item.
  int x = A[threadIdx.x];
  A[threadIdx.x] = x * x;
}

__global__ void vcube_(int *A) {  // NOLINT
  // Kernel computes vsqr for one  data-item.
  int x = A[threadIdx.x];
  A[threadIdx.x] = x * x * x;
}

__global__ void vadd(const int *A, const int *B, int *C) {
  // Kernel computes vsqr for one  data-item.
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

__global__ void fused_sqr_cub_add(const int *A, const int *B, int *C) {
  int i = threadIdx.x;
  int x = A[i];
  int y = B[i];
  C[i] = (x * x) + (y * y * y);
}

__global__ void print_hello_world() {
  // Print hello world.
  // Unsure how this is working, because printf is code that will run on host.
  // Does this mean device can call functions that execute on the host?
  printf("Hello World.\n");
}

__global__ void scalar_init(int *A) {
  dim_t id = blockIdx.x * blockDim.x + threadIdx.x;
  A[id] = static_cast<int>(id);
}

__global__ void matrix_square_v1(const int *A, dim_t N, int *B) {
  dim_t id = blockIdx.x * blockDim.x + threadIdx.x;
  for (dim_t j = 0; j < N; ++j) {
    for (dim_t k = 0; k < N; ++k) {
      B[id * N + j] += A[id * N + k] * A[k * N + j];
    }
  }
}

__global__ void matrix_square_v2(const int *A, dim_t N, int *B) {
  dim_t id = blockIdx.x * blockDim.x + threadIdx.x;
  dim_t i = id / N;
  dim_t j = id % N;
  for (dim_t k = 0; k < N; ++k) {
    B[i * N + j] += A[i * N + k] * A[k * N + j];
  }
}

__global__ void warp_branch_paths(int *A, dim_t size) {
  dim_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id % 2) {
    A[id] = id;
  } else {
    A[id] = size * size;
  }
  A[id]++;
}
