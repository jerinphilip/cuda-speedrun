#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstdio>

#include "fcl/error.h"

void gpuAssert(hipError_t code, const char *file, int line,
               bool abort /* = true*/) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}

__device__ void cdpAssert(hipError_t code, const char *file, int line,
                          bool abort /* = true*/) {
  if (code != hipSuccess) {
    printf("GPU kernel assert: %s %s %d\n", hipGetErrorString(code), file,
           line);
    if (abort) assert(0);
  }
}
