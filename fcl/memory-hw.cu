#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdio>
#include <vector>

#include "buffer.h"

#define N 1000

__global__ void vsqr_(int *A) {  // NOLINT
  // Kernel computes vsqr for one  data-item.
  int x = A[threadIdx.x];
  A[threadIdx.x] = x * x;
}

__global__ void vcube_(int *A) {  // NOLINT
  // Kernel computes vsqr for one  data-item.
  int x = A[threadIdx.x];
  A[threadIdx.x] = x * x * x;
}

__global__ void vadd(const int *A, const int *B, int *C) {
  // Kernel computes vsqr for one  data-item.
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

__global__ void fused_sqr_cub_add(const int *A, const int *B, int *C) {
  int i = threadIdx.x;
  int x = A[i], y = B[i];  // NOLINT
  C[i] = x * x + y * y * y;
}

std::vector<int> generate(size_t size) {
  std::vector<int> data(size);
  for (size_t i = 0; i < size; i++) {
    data[i] = i;
  }
  return data;
}

int main() {
  // Copy data to a GPU memory buffer.
  std::vector<int> first = generate(N);
  std::vector<int> second = generate(N);

  Buffer<int> g_first(first.data(), first.size());
  Buffer<int> g_second(second.data(), second.size());

  auto pipelined = [&]() {
    Buffer<int> g_result(N);
    vsqr_<<<1, N>>>(g_first.data());
    vcube_<<<1, N>>>(g_second.data());
    vadd<<<1, N>>>(g_first.data(), g_second.data(), g_result.data());
    return g_result;
  };

  auto fused = [&]() {
    Buffer<int> g_result(N);
    fused_sqr_cub_add<<<1, N>>>(g_first.data(), g_second.data(),
                                g_result.data());
    return g_result;
  };

  // Buffer<int> g_result = pipelined();
  Buffer<int> g_result = fused();

  Buffer<int> result = g_result.cpu();
  for (size_t i = 0; i < result.size(); i++) {  // NOLINT
    int x = first[i], y = second[i];            // NOLINT
    if (result[i] != x * x + y * y * y) {
      fprintf(stderr, "Mismatch found.\n");
      std::abort();
    };
    printf("%d\n", result[i]);
  }
  return 0;
}
