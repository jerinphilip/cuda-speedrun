#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#incldue "buffer.h"

#include <cstdio>
#include <vector>

#define N 1000

// https://stackoverflow.com/a/14038590/4565794
#define gpuErrchk(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}

__global__ void vsqr_(int *A) {  // NOLINT
  // Kernel computes vsqr for one  data-item.
  int x = A[threadIdx.x];
  A[threadIdx.x] = x * x;
}

__global__ void vcube_(int *A) {  // NOLINT
  // Kernel computes vsqr for one  data-item.
  int x = A[threadIdx.x];
  A[threadIdx.x] = x * x * x;
}

__global__ void vadd(const int *A, const int *B, int *C) {
  // Kernel computes vsqr for one  data-item.
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

__global__ void fused_sqr_cub_add(const int *A, const int *B, int *C) {
  int i = threadIdx.x;
  int x = A[i], y = B[i];  // NOLINT
  C[i] = x * x + y * y * y;
}

std::vector<int> generate(size_t size) {
  std::vector<int> data(size);
  for (size_t i = 0; i < size; i++) {
    data[i] = i;
  }
  return data;
}

int main() {
  // Copy data to a GPU memory buffer.
  std::vector<int> first = generate(N);
  std::vector<int> second = generate(N);

  GPUBuffer<int> g_first(first.data(), first.size());
  GPUBuffer<int> g_second(second.data(), second.size());

  auto pipelined = [&]() {
    GPUBuffer<int> g_result(N);
    vsqr_<<<1, N>>>(g_first.data());
    vcube_<<<1, N>>>(g_second.data());
    vadd<<<1, N>>>(g_first.data(), g_second.data(), g_result.data());
    return g_result;
  };

  auto fused = [&]() {
    GPUBuffer<int> g_result(N);
    fused_sqr_cub_add<<<1, N>>>(g_first.data(), g_second.data(),
                                g_result.data());
    return g_result;
  };

  // GPUBuffer<int> g_result = pipelined();
  GPUBuffer<int> g_result = fused();

  std::vector<int> result = g_result.cpu();
  for (size_t i = 0; i < result.size(); i++) {  // NOLINT
    int x = first[i], y = second[i];            // NOLINT
    if (result[i] != x * x + y * y * y) {
      fprintf(stderr, "Mismatch found.\n");
      std::abort();
    };
    printf("%d\n", result[i]);
  }
  return 0;
}
