#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

__device__ volatile int secondkernel, preprocessingdone, perthread;

__device__ void preprocess() {
	int a[10] = {0};
	for (int ii = 0; ii < 10000; ++ii)
		a[ii % 10]++;
}
__global__ void K1(volatile int *perthread) {
	preprocess();
	__syncthreads();

	preprocessingdone = 1;
	*perthread = 100;	// some number.

	if (secondkernel) {
		*perthread /= 2;
	}
	//if (threadIdx.x == 0) printf("perthread = %d\n", *perthread);
}
__global__ void K2(volatile int *perthread) {

	if (preprocessingdone)
		;	// do nothing.
	else {
		secondkernel = 1;
		*perthread = 100 / 2;
	}
}

__global__ void Kinit() {
	secondkernel = 0;
	preprocessingdone = 0;
}
int main() {
    srand(time(NULL));
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    volatile int *perthread;
    hipMalloc((int **)&perthread, sizeof(int));

    for (int ii = 0; ii < 10; ++ii) {

	Kinit<<<1, 1>>>();
	hipDeviceSynchronize();

	K1<<<1, 64, 0, s1>>>(perthread);

	if (rand() % 2) {
		K2<<<1, 64, 0, s2>>>(perthread);
		printf("two kernels: ");
	} else
		printf("one kernel: ");

	hipDeviceSynchronize();

	int hpt;
	hipMemcpy(&hpt, (int *)perthread, sizeof(int), hipMemcpyDeviceToHost);
	printf("per thread = %d\n", hpt);
    }

    return 0;
}
